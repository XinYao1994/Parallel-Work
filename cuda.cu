#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <string.h>
#include <hip/hip_runtime.h>
//#pragma comment(lib, "cudart.lib")
#define LA 1024
#define LB 1024
#define LC 1024
#define num 50

typedef struct{
  int *m;
  int a,b;
}Matrix;

hipError_t error;
int block_size = 16;

bool init(){
  int count;
  int devID = 0;
  hipDeviceProp_t deviceProp;
  hipGetDeviceCount(&count);
  if(count == 0){
     printf("no device, exit\n");
     return false;
  }
  error = hipGetDevice(&devID);
  error = hipGetDeviceProperties(&deviceProp, devID);
  if(error != hipSuccess){
     printf("device not perpared,exit\n");
     return false;
  }
  block_size = (deviceProp.major < 2) ? 16 : 32;
  return true;
}

int Myatoi(char *c){
  int sum = 0;
  while(*c != '\0'){
    sum *= 10;
    sum += (*c - '0');
    c++;
  }
  return sum;
}

Matrix *a,*b,*c;

void allocMm(Matrix **m, int x,int y,int init){
  *m = (Matrix *)malloc(sizeof(Matrix));
  (*m)->a = x;
  (*m)->b = y;
  (*m)->m = (int *)malloc(sizeof(int)*x*y);
  if(!init){
     memset((*m)->m,0,sizeof(int)*x*y);
     return ;
  }
  int i,j;
  for(i=0;i<x;i++)
    for(j=0;j<y;j++)
       *((*m)->m+i*y+j) = rand()%num;
}

template <int BLOCK_SIZE> __global__ void
matrixMulCuda(int *c, int *a, int *b, int wA, int wB){
  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  
  int aBegin = wA * BLOCK_SIZE * by;
  int aEnd = aBegin + wA - 1;
  int aStep  = BLOCK_SIZE;
  int bBegin = BLOCK_SIZE * bx;
  int bStep  = BLOCK_SIZE * wB;
  int Cadd = 0;
  for (int i = aBegin, j = bBegin;i <= aEnd; i += aStep, j += bStep)
  {
      __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
      __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];
      As[ty][tx] = a[i + wA * ty + tx];
      Bs[ty][tx] = b[j + wB * ty + tx];
      __syncthreads();
      for (int k = 0; k < BLOCK_SIZE; ++k)
      {
            Cadd += As[ty][k] * Bs[k][tx];
      }
      __syncthreads();
    }
    int w = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
    c[w + wB * ty + tx] = Cadd;
}

void matrixMul(Matrix *a, Matrix *b, Matrix *c, int block_size){
  int *d_a, *d_b, *d_c;
  hipMalloc((void **)&d_a, sizeof(int)*a->a*a->b);
  hipMalloc((void **)&d_b, sizeof(int)*b->a*b->b);
  hipMalloc((void **)&d_c, sizeof(int)*c->a*c->b);
  error = hipMemcpy(d_a, a->m, sizeof(int)*a->a*a->b, hipMemcpyHostToDevice);
  error = hipMemcpy(d_b, b->m, sizeof(int)*b->a*b->b, hipMemcpyHostToDevice);
  if(error != hipSuccess) {
     printf("copy failed from host to device, exit\n");
     exit(0);
  }
  dim3 threads(block_size, block_size);
  dim3 grid(b->b/threads.x, a->a/threads.y);
  if(block_size==16) 
     matrixMulCuda<16><<< grid, threads >>>(d_c, d_a, d_b, a->b, b->b);
  else
     matrixMulCuda<32><<< grid, threads >>>(d_c, d_a, d_b, a->b, b->b);
  error = hipMemcpy(c->m, d_c, sizeof(int)*c->a*c->b, hipMemcpyDeviceToHost);
  if(error != hipSuccess)  {
     printf("copy failed from device to host, exit\n");
     exit(0);
  }
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
}

int main(int argc, char **argv){
  srand(time(0));
  if(!init()) exit(0);
  int la, lb, lc;
  if(argc==2){
    la = lb = lc = atoi(*(argv+1));
  }
  else if(argc==4){
    la = Myatoi(*(argv+1));
    lb = Myatoi(*(argv+2));
    lc = Myatoi(*(argv+3));
  }
  else if(argc==1){
    printf("use 1024, 1024, 1024\n");
    la = LA; lb = LB; lc = LC;
  }
  else{
    if(argc>1) printf("argc error,use 1024, 1024, 1024\n");
    la = LA; lb = LB; lc = LC;
  }
  if(!la%block_size || !lb%block_size || !lc%block_size){
    printf("set matrix's parameter%(%d) = 0\n",block_size);
    return;
  }
  //alloc the matrix
  allocMm(&a, la, lb, 1);
  allocMm(&b, lb, lc, 1);
  allocMm(&c, la, lc, 0);    
  matrixMul(a, b, c, block_size);
  return 0;
}
